
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define f_size sizeof(float)
#define i_size sizeof(int)

//Opens a matrix store and represents it using Compressed Sparse Row format
void matrix_read(int **row_pointer, int **column_index, float **values, const char *storename, int *nrows, int *ncols, int *nvals) {

    FILE *store = fopen(storename, "r");
    
    if (store == NULL) {
    
        fprintf(stdout, "File error!\n");
        exit(0);
    }
    
    //first line represents no. rows, columns, nnz values
    fscanf(store, "%d %d %d\n", nrows, ncols, nvals);
    
    int *row_pointer_temp = (int*) malloc( i_size * (*nrows + 1) );
    int *column_index_temp = (int*) malloc( i_size * (*nvals) );
    float *values_temp = (float*) malloc( f_size * (*nvals) );
    
    //count appearances of each row for indices of row_pointer array
    int *r_occ= (int *) malloc(i_size * (*nrows));
    
    for (int i = 0; i < *nrows; i++) {
        r_occ[i] = 0;
    }
    
    int r;
    int c;
    float v;
    
    //C format
    while (fscanf(store, "%d %d %f\n", &r, &c, &v) != EOF) {
        r = r-1;
        c = c-1;
        r_occ[r] = r_occ[r] + 1;
    }
    
    //Fill row_pointer
    int idx = 0;
    
    for (int i = 0; i < *nrows; i++) {
    
        row_pointer_temp[i] = idx;
        idx += r_occ[i];
    }
    
    row_pointer_temp[*nrows] = *nvals;
    free(r_occ);
    
    //Rewind to beginning of file store
    rewind(store);
    
    //Capture column indices and values
    for (int i = 0; i < *nvals; i++) {
    
        column_index_temp[i] = -1;
    }
    
    fscanf(store, "%d %d %d\n", nrows, ncols, nvals);
    int i = 0;
    
    while (fscanf(store, "%d %d %f\n", &r, &c, &v) != EOF) {
    
        r = r - 1;
        c = c - 1;
        
        //Get right index using row information and index i
        while (column_index_temp[i + row_pointer_temp[r]] != -1) {
        
            i++;
        }
        
        column_index_temp[i + row_pointer_temp[r]] = c;
        values_temp[i + row_pointer_temp[r]] = v;
        i = 0;
    }
    
    fclose(store);
    
    *row_pointer = row_pointer_temp;
    *column_index = column_index_temp;
    *values = values_temp;
}

//SpMV kernel using CSR 
__global__ void csr_spMV(const int *row_pointer, const int *column_index, const float *values, const int nrows, const float *x, float *y) {
    // Uses a grid-stride loop to perform dot product
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int a = i ; a < nrows; a = a + (blockDim.x * gridDim.x)) {
    
        float prod = 0;
        const int row_start = row_pointer[a];
        const int row_end = row_pointer[a + 1];
        
        for (int b = row_start; b < row_end; b++) {
        
            prod += values[b] * x[ column_index[b] ];
            
        }
        
        y[a] = (float)prod;
    }
 }
 
int main(int argc, const char * argv[]) {
    if (argc != 5) {
    
        fprintf(stdout, "Invalid, enter:\n1. number of threads, 2. number of repetitions, 3. print mode (1 or 2), 4. test storename\n");
        exit(0);
    }
    
    int *row_pointer, *column_index, nrows, ncols, nvals, numSMs;;
    float *values;
    
    int nthread = atoi(argv[1]);
    int nrepeat = atoi(argv[2]);
    int pmode = atoi(argv[3]);
    const char *storename = argv[4];
    
    matrix_read(&row_pointer, &column_index, &values, storename, &nrows, &ncols, &nvals);
    
    float *x = (float *) malloc(nrows * f_size);
    float *y = (float *) malloc(nrows * f_size);
    
    for (int i = 0; i < nrows; i++) {
    
        x[i] = 1.0;
        y[i] = 0.0;
    }
    
    if (pmode == 1) {
        // val store
        fprintf(stdout, "Value array:\n");
        
        for (int j = 0; j < nvals; j++) {
        
            fprintf(stdout, "%.2f ", values[j]);
            
        }
        
        // Column Indices Array
        fprintf(stdout, "\n\nColumn Indices Array:\n");
        
        for (int j = 0; j < nvals; j++) {
        
            fprintf(stdout, "%d ", column_index[j]);
            
        }
        
        // row pointer store
        fprintf(stdout, "\n\nRow pointer array:\n");
        
        for (int j = 0; j < (nrows + 1); j++) {
        
            fprintf(stdout, "%d ", row_pointer[j]);
            
        }
        
        fprintf(stdout, "\n\nDense vector:\n");
        
        for (int j = 0; j < nrows; j++) {
        
            fprintf(stdout, "%.1f ", x[j]);
            
        }
        
        fprintf(stdout, "\n\nReturned vector:\n");
    }
    
    // Memory allocation
    int *device_row_pointer, *device_column_index;
    float *device_vals, *device_Vx, *device_Vy;
    
    hipMalloc((void**) &device_row_pointer, (nrows + 1) * i_size);
    hipMalloc((void**) &device_column_index, nvals * i_size);
    hipMalloc((void**) &device_vals, nvals * f_size);
    hipMalloc((void**) &device_Vx, nrows * f_size);
    hipMalloc((void**) &device_Vy, nrows * f_size);
    
    //SMs in device
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    
    // move data from host to device
    hipMemcpy(device_row_pointer, row_pointer, (nrows + 1) * i_size, hipMemcpyHostToDevice);
    hipMemcpy(device_column_index, column_index, nvals * i_size, hipMemcpyHostToDevice);
    hipMemcpy(device_vals, values, nvals * f_size, hipMemcpyHostToDevice);
    
    // Time kernel
    float elapsed_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    for (int i = 0; i < nrepeat; i++) {
    
        hipMemcpy(device_Vx, x, nrows * f_size, hipMemcpyHostToDevice);
        hipMemcpy(device_Vy, y, nrows * f_size, hipMemcpyHostToDevice);
        
        //kernel
        csr_spMV<<<32 * numSMs, nthread>>>(device_row_pointer, device_column_index, device_vals, nrows, device_Vx, device_Vy);
        
        //move result from device to host
        hipMemcpy(y, device_Vy, nrows * f_size, hipMemcpyDeviceToHost);
        
        for (int i = 0; i < nrows; i++) {
        
            x[i] = (float)y[i];
            y[i] = 0.0;
        }
    }
    
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    
    //Resultant vector
    if (pmode == 1 || pmode == 2) {
    
        for (int j = 0; j < nrows; j++) {
        
            fprintf(stdout, "%.2f\n", x[j]);
        }
        fprintf(stdout, "\n");
    }
    
    // Print time taken
    printf("\nParallel Run time:  %.4f ms\n", elapsed_time);
    
    // Free memory
    hipFree(device_row_pointer);
    hipFree(device_column_index);
    hipFree(device_vals);
    hipFree(device_Vx);
    hipFree(device_Vy);
    
    free(row_pointer);
    free(column_index);
    free(values);
    
    return 0;
}
